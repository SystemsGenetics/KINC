
#include <hip/hip_runtime.h>

// #include "linalg.cu"



/*!
 * Compute the initial labels for a gene pair in an expression matrix. Samples
 * with missing values and samples that are outside the expression thresholds are
 * labeled as such, all other samples are labeled as cluster 0. The number of
 * clean samples is returned.
 *
 * @param x
 * @param y
 * @param sampleSize
 * @param minExpression
 * @param maxExpression
 * @param labels
 */
__device__
int fetchPair(
    const float *x,
    const float *y,
    int sampleSize,
    float minExpression,
    float maxExpression,
    char *labels)
{
    // label the pairwise samples
    int N = 0;

    for ( int i = 0; i < sampleSize; ++i )
    {
        // label samples with missing values
        if ( isnan(x[i]) || isnan(y[i]) )
        {
            labels[i] = -9;
        }

        // label samples which are below the minimum expression threshold
        else if ( x[i] < minExpression || y[i] < minExpression )
        {
            labels[i] = -6;
        }

        // label samples which are above the maximum expression threshold
        else if ( x[i] > maxExpression || y[i] > maxExpression )
        {
            labels[i] = -6;
        }

        // label any remaining samples as cluster 0
        else
        {
            N++;
            labels[i] = 0;
        }
    }

    // return number of clean samples
    return N;
}
