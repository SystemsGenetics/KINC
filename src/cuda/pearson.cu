
#include <hip/hip_runtime.h>






/*!
 * Compute the Pearson correlation of a cluster in a pairwise data array.
 *
 * @param x
 * @param y
 * @param labels
 * @param sampleSize
 * @param cluster
 * @param minSamples
 */
__device__
float Pearson_computeCluster(
   const float *x,
   const float *y,
   const char *labels,
   int sampleSize,
   char cluster,
   int minSamples)
{
   // compute intermediate sums
   int n = 0;
   float sumx = 0;
   float sumy = 0;
   float sumx2 = 0;
   float sumy2 = 0;
   float sumxy = 0;

   for ( int i = 0; i < sampleSize; ++i )
   {
      if ( labels[i] == cluster )
      {
         float x_i = x[i];
         float y_i = y[i];

         sumx += x_i;
         sumy += y_i;
         sumx2 += x_i * x_i;
         sumy2 += y_i * y_i;
         sumxy += x_i * y_i;

         ++n;
      }
   }

   // compute correlation only if there are enough samples
   float result = NAN;

   if ( n >= minSamples )
   {
      result = (n*sumxy - sumx*sumy) / sqrt((n*sumx2 - sumx*sumx) * (n*sumy2 - sumy*sumy));
   }

   return result;
}






/*!
 * Compute the correlation of each cluster in a pairwise data array. The data array
 * should only contain the clean samples that were extracted from the expression
 * matrix, while the labels should contain all samples.
 *
 * @param x
 * @param y
 * @param sampleSize
 * @param clusterSize
 * @param labels
 * @param minSamples
 * @param correlations
 */
__device__
void Pearson_compute(
   const float *x,
   const float *y,
   int sampleSize,
   char clusterSize,
   const char *labels,
   int minSamples,
   float *correlations)
{
   for ( char k = 0; k < clusterSize; ++k )
   {
      correlations[k] = Pearson_computeCluster(
         x, y,
         labels,
         sampleSize,
         k,
         minSamples
      );
   }
}
